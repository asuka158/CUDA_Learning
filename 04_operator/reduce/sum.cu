#include "hip/hip_runtime.h"
#define CEIL(a, b) ((a + b - 1) / (b))
#define FLOAT4(value) (reinterpret_cast<float4*>(&(value))[0])

dim3 block_size(BLOCK_SIZE);
dim3 grid_size(CIEL(N, BLOCK_SIZE));

__global__ void reduce_v1(const float* input, float* output, int N)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if(idx < N) atomicAdd(output, input[idx]);
}

__global__ void reduce_v2(const float* input, float* output, int N)
{
    int tid = threadIdx.x;
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    
    __shared__ float input_s[BLOCK_SIZE];

    input_s[tid] = (idx < N) ? input[idx] : 0.0f;
    __syncthreads();

    for(int offset = blockDim.x >> 1; offset > 0; offset >>= 1)
    {
        if(tid < offset) input_s[tid] += input_s[tid + offset];
        __syncthreads();
    }

    if(tid == 0) atomicAdd(output, input_s[0]);
}

__global__ void reduce_v3(float* d_x, float* d_y, const int N)
{
    __shared__ float s_y[32];

    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int warpId = threadIdx.x / warpSize;
    int laneId = threadIdx.x & (warpSize - 1);

    float val = (idx < N) ? d_x[idx] : 0.0f;
    
    #pragma unroll
    for(int offset = warpSize >> 1; offset > 0; offset >>= 1) 
        val += __shfl_down_sync(0xFFFFFFFF, val, offset);

    if(laneId == 0) s_y[warpId] = val;
    __syncthreads();

    if(warpId == 0)
    {
        int warpNum = blockDim.x / warpSize;
        val = (laneId < warpNum) ? s_y[laneId] : 0.0f;
        for(int offset = warpSize >> 1; offset > 0; offset >>= 1) 
            val += __shfl_down_sync(0xFFFFFFFF, val, offset);
        if(laneId == 0) atomicAdd(d_y, val);
    }
}

__global__ void reduce_v4(float* d_x, float* d_y, const int N)
{
    __shared__ float s_y[32];

    int idx = (blockDim.x * blockIdx.x + threadIdx.x) * 4;
    int warpId = threadIdx.x / warpSize;
    int laneId = threadIdx.x & (warpSize - 1);

    float val = 0.0f;
    if(idx < N) 
    {
        float4 tmp_x = FLOAT4(d_x[idx]);
        val += tmp_x.x;
        val += tmp_x.y;
        val += tmp_x.z;
        val += tmp_x.w;
    }

    #pragma unroll
    for(int offset = warpSize >> 1; offset > 0; offset >>= 1) 
        val += __shfl_down_sync(0xFFFFFFFF, val, offset);

    if(laneId == 0) s_y[warpId] = val;
    __syncthreads();

    if(warpId == 0)
    {
        int warpNum = blockDim.x / warpSize;
        val = (laneId < warpNum) ? s_y[laneId] : 0.0f;
        for(int offset = warpSize >> 1; offset > 0; offset >>= 1) 
            val += __shfl_down_sync(0xFFFFFFFF, val, offset);
        if(laneId == 0) atomicAdd(d_y, val);
    }
}